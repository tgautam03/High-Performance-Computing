#include "hip/hip_runtime.h"
// Forward declaration
__global__ void addKernel(float *a, float *b, float *c, int N);

void add(Vector v1, Vector v2, Vector v3) 
{
    assert (v1.device == false && v2.device == false && v3.device == false);
    assert (v1.len == v2.len);
    assert (v1.len == v3.len);

    for (int i = 0; i < v1.len; i++)
        v3.ptr[i]=v1.ptr[i] + v2.ptr[i];
}


void addDevice(Vector &v1, Vector &v2, Vector &v3, int numThrds)
{
    assert (v1.device == true && v2.device == true && v3.device == true);
    assert (v1.len == v2.len);
    assert (v1.len == v3.len);

    // addDevice(d_v1, d_v2, d_v3, 256);
    int thr_per_blk = numThrds;
    int blk_in_grid = ceil( float(v1.len) / thr_per_blk );

    // Launch kernel
    addKernel<<< blk_in_grid, thr_per_blk >>>(v1.ptr, v2.ptr, v3.ptr, v1.len);
}


// Kernel
__global__ void addKernel(float *a, float *b, float *c, int N)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < N) c[id] = a[id] + b[id];
}
