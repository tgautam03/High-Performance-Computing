#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

void add(Vector v1, Vector v2, Vector v3) {
    assert (v1.device == false && v2.device == false && v3.device == false);
    assert (v1.len == v2.len);
    assert (v1.len == v3.len);

    for (int i = 0; i < v1.len; i++)
        v3.put(v1[i] + v2[i], i);
}