#include <iostream>
#include <stdlib.h>
#include <assert.h>


// Randomly initialise a vector
void rand_init(Vector h_v) 
{
    assert (h_v.device == false);
    for (int i = 0; i < h_v.len; i++)
    {
        float r1 = rand() % 100;
        float r2 = rand() % 10;

        h_v.ptr[i] = r1/r2;
    }
}

// Initialise a vector with specific value
void val_init(Vector h_v, float val) 
{
    assert (h_v.device == false);
    for (int i = 0; i < h_v.len; i++)
        h_v.ptr[i] = val;
}

// Print the vector
void print(Vector h_v, string name)
{
    assert (h_v.device == false);
    std::cout << "Vector " << name << ": " ;

    for (int i = 0; i < h_v.len; i++)
        std::cout << h_v.ptr[i] << " ";
    std::cout << "\n";
}

// Move data to device
Vector toDevice(Vector h_v, bool del=false)
{
    assert (h_v.device == false);

    Vector d_v(h_v.len, true);
    hipMemcpy(d_v.ptr, h_v.ptr, h_v.len*sizeof(float), hipMemcpyHostToDevice);

    if (del==true)
        h_v.free();

    return d_v;
}

// Move data to Host 
Vector toHost(Vector d_v, bool del=true)
{
    assert (d_v.device == true);

    Vector h_v(d_v.len, false);
    hipMemcpy(h_v.ptr, d_v.ptr, d_v.len*sizeof(float), hipMemcpyDeviceToHost);

    if (del==true)
        d_v.free();

    return h_v;
}