#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>


using namespace std;

class Vector 
{
    public:
    // Length attribute
    int len = 0;
    // Pointer to the vector
    float *ptr = 0;
    // device
    bool device = false;

    // Constructor to setup vector pointer
    Vector(int, bool); 

    // Free up the memory
    void free();
};


// Constructor to setup vector pointer
Vector::Vector(int len_, bool device_=false) 
{
    Vector::len = len_;
    Vector::device = device_;

    if (device_ == false)
        Vector::ptr = new float[len_];
    else
    {
        hipError_t err = hipMalloc((void**)&(Vector::ptr), (Vector::len)*sizeof(float));

        if (err != hipSuccess)
        {
            std::cout << hipGetErrorString(err) << " in " 
            << __FILE__ << " at " << __LINE__ << "\n";

            exit(EXIT_FAILURE);
        }
        else
            std::cout << "Memory Allocation on Device success! \n";
    }
}


void Vector::free() {
    if (Vector::device == false)
    {
        delete[] Vector::ptr;
        std::cout << "Freed host memory! \n";
    }
    else
    {
        hipFree(Vector::ptr);
        std::cout << "Freed device memory! \n";
    }
}