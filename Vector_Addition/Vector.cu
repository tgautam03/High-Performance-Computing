#include <iostream>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>


using namespace std;

class Vector 
{
    public:
    // Length attribute
    int len = 0;
    // Pointer to the vector
    float *ptr = 0;
    // device
    bool device = false;

    // Constructor to setup vector pointer
    Vector(int len, bool device=false) 
    {
        Vector::len = len;
        Vector::device = device;

        if (device == false)
            Vector::ptr = new float[len];
        else
            hipMalloc((void**)&(Vector::ptr), (Vector::len)*sizeof(float));
    }

    // Get value at a specific index
    float operator[](int idx);

    // Assign value at a specific index
    void put(float val, int idx);

    // Free up the memory
    void free();
};

float Vector::operator[](int idx) {
    return Vector::ptr[idx];
}

void Vector::put(float val, int idx) {
    Vector::ptr[idx] = val;
}

void Vector::free() {
    if (Vector::device == false)
    {
        delete[] Vector::ptr;
        std::cout << "Freed host memory! \n";
    }
    else
    {
        hipFree(Vector::ptr);
        std::cout << "Freed device memory! \n";
    }
}