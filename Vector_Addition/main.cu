#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
using namespace std;

#include "Vector.cu"
#include "utils.cu"
#include "vecAdd.cu"

int main() {
    // Length of an array
    int len = 0;
    std::cout <<"Enter the length of vector: ";
    std::cin >> len;

    // Defining Vectors
    Vector v1(len), v2(len), v3(len);

    // Initialising vectors
    rand_init(v1);
    rand_init(v2);
    val_init(v3,0);

    print(v1, "v1");
    print(v2, "v2");
    print(v3, "v3");


    // Moving to GPU and not deleting the data in RAM
    Vector d_v1 = toDevice(v1, false);
    Vector d_v2 = toDevice(v2, false);
    Vector d_v3 = toDevice(v3, false);

    // CPU sum
    add(v1, v2, v3);
    print(v3, "v3");

    // GPU sum
    int numThrds = 256;
    addDevice(d_v1, d_v2, d_v3, numThrds);
    
    // Moving back from GPU
    Vector h_v3 = toHost(d_v3);

    print(h_v3, "h_v3");
    
    return 0;
}