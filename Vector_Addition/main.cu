#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
using namespace std;

#include "Vector.cu"
#include "utils.cu"
// #include "vecAdd.cu"

int main() {
    // Length of an array
    int len = 0;
    std::cout <<"Enter the length of vector: ";
    std::cin >> len;

    Vector v1(len), v2(len), v3(len);
    // initialise(v1, len); initialise(v2, len); initialise(v3, len);

    // Initialising vectors
    rand_init(v1);
    rand_init(v2);
    val_init(v3,0);

    print(v1, "v1");
    print(v2, "v2");
    print(v3, "v3");


    // Moving to GPU
    Vector d_v1 = toDevice(v1, false);
    Vector d_v2 = toDevice(v2, false);
    Vector d_v3 = toDevice(v3, false);

    // CPU sum
    // add(v1, v2, v3);
    // print(v3, "v3");

    // GPU sum
    int numThrds = 256;
    // int thr_per_blk = numThrds;
    // int blk_in_grid = ceil( float(v1.len) / thr_per_blk );

    // // Launch kernel
    // addKernel<<< blk_in_grid, thr_per_blk >>>(d_v1.ptr, d_v2.ptr, d_v3.ptr, d_v1.len);
    // addDevice(d_v1, d_v2, d_v3, numThrds);
    
    // Moving back from GPU
    // Vector h_v3 = toHost(d_v3);

    // print(h_v3, "h_v3");

    return 0;
}